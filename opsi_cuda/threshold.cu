
#include <hip/hip_runtime.h>
extern "C"{
__global__ void applyFilter(unsigned char *inputRed, unsigned char *inputGreen, unsigned char *inputBlue,
                             unsigned char *outputChannel,
                             const unsigned int width, const unsigned int height,
                             int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2) {
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;

   if(row < height && col < width) {
        uchar3 v = make_uchar3(inputRed[row + col * width], inputGreen[row + col * width], inputBlue[row + col * width]);
        if (v.x >= lbc0 && v.x <= ubc0 && v.y >= lbc1 && v.y <= ubc1 && v.z >= lbc2 && v.z <= ubc2)
        outputChannel[row + col * width] = 255;
        else
        outputChannel[row + col * width] = 0;
    }
}
}